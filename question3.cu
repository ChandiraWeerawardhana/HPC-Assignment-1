#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void vecAdd3(float* A, float* B, float* C, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main(void) {
    const int threadsPerBlock = 1024;
    const int N = 1024 * 100000;
    size_t size = N * sizeof(float);
    float *a, *b, *c;
    float *dev_a, *dev_b, *dev_c;

    a = (float*)malloc(size);
    b = (float*)malloc(size);
    c = (float*)malloc(size);

    for (int i = 0; i < N; i++) {
        a[i] = rand() % 1000;
        b[i] = rand() % 1000;
    }

    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);

    hipEvent_t startUpload, endUpload, startKernel, endKernel, startDownload, endDownload;
    hipEventCreate(&startUpload);
    hipEventCreate(&endUpload);
    hipEventCreate(&startKernel);
    hipEventCreate(&endKernel);
    hipEventCreate(&startDownload);
    hipEventCreate(&endDownload);

    //to measure upload time (Host to Device)
    hipEventRecord(startUpload);
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    hipEventRecord(endUpload);
    //to measure kernel execution time
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    hipEventRecord(startKernel);
    vecAdd3<<<numBlocks, threadsPerBlock>>>(dev_a, dev_b, dev_c, N);
    hipEventRecord(endKernel);

    //to measure download time (Device to Host)
    hipEventRecord(startDownload);
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    hipEventRecord(endDownload);

    hipEventSynchronize(endUpload);
    hipEventSynchronize(endKernel);
    hipEventSynchronize(endDownload);

    float timeUpload, timeKernel, timeDownload;
    hipEventElapsedTime(&timeUpload, startUpload, endUpload);
    hipEventElapsedTime(&timeKernel, startKernel, endKernel);
    hipEventElapsedTime(&timeDownload, startDownload, endDownload);

    float timeTotal = timeUpload + timeKernel + timeDownload;

    printf("Upload Time (Host to Device): %.4f ms\n", timeUpload);
    printf("Kernel Execution Time: %.4f ms\n", timeKernel);
    printf("Download Time (Device to Host): %.4f ms\n", timeDownload);
    printf("Total Time Taken: %.4f seconds\n", timeTotal/1000);

    for (int i = 0; i < 10; i++) {
        printf("%.1f + %.1f = %.1f\n", a[i], b[i], c[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    free(a);
    free(b);
    free(c);

    hipEventDestroy(startUpload);
    hipEventDestroy(endUpload);
    hipEventDestroy(startKernel);
    hipEventDestroy(endKernel);
    hipEventDestroy(startDownload);
    hipEventDestroy(endDownload);

    return 0;
}
