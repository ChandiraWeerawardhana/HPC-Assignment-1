#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void vecAdd(float* A, float* B, float* C, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main(void) {
    const int N = 1024 * 10;
    float *a, *b, *c;
    float *dev_a, *dev_b, *dev_c;

    size_t size = N * sizeof(float);

    a = (float*)malloc(size);
    b = (float*)malloc(size);
    c = (float*)malloc(size);

    for (int i = 0; i < N; i++) {
        a[i] = rand() % 1000;
        b[i] = rand() % 1000;
    }

    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    bool error = false;

    for (int threads = 1; threads <= 2048; threads++) {

        vecAdd<<<1, threads>>>(dev_a, dev_b, dev_c, N);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            if (!error) {
                printf("\nKernel launch failed at %d threads per block.\n", threads);
                printf("Therefore, the maximum supported threads per block is %d.\n", threads - 1);
                error = true;
            }
            break;
        }
    }

    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

    printf("\nSample output for N=%d elements:\n", N);
    for (int i = 0; i < 10; i++) {
        printf("%.1f + %.1f = %.1f\n", a[i], b[i], c[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    free(a);
    free(b);
    free(c);

    return 0;
}
